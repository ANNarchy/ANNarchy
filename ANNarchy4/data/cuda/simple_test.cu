#include "hip/hip_runtime.h"
#include "simple_test.h"
#include <stdio.h>
#include <iostream>

__global__ void helloCudaKernel()
{
	printf("Hello from your device :-)");
}

void helloCuda()
{
	hipSetDevice(0);
	std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
	
	printf("test ... \n");
	helloCudaKernel<<<1,1>>>();
	
	std::cout << hipGetErrorString(hipGetLastError()) << std::endl;
	hipDeviceSynchronize(); // synchronize the printf
}