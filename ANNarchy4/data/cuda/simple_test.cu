#include "hip/hip_runtime.h"
#include "simple_test.h"
#include <stdio.h>
#include <iostream>
#include <vector>
#include <omp.h>

__global__ void helloCudaKernel()
{
	printf("Hello from your device :-) \n");
}

void helloCuda()
{
	hipSetDevice(0);
	
	helloCudaKernel<<<1,1>>>();
	hipDeviceSynchronize(); // synchronize the printf
}

template<class T, unsigned int blockSize>
__global__ void
weightReduce(
		T *pr,		// neurons preynaptic layer
		T *w,		// weights matrix per neuron
		int *idx,	// index matrix per neuron
		int c,		// number of connections
		T *result	// write back result
	  ) {

	unsigned int tid = threadIdx.x;
    unsigned int i = tid;

	extern T __shared__ sdata[];
	T mySum = 0.0;

	while(i < c) {
		mySum += pr[idx[i]] * w[i];

		i+= blockSize;
	}

    sdata[tid] = mySum;
    __syncthreads();

    // do reduction in shared mem
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] = mySum = mySum + sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] = mySum = mySum + sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid <  64) { sdata[tid] = mySum = mySum + sdata[tid +  64]; } __syncthreads(); }

    if (tid < 32)
    {
        // now that we are using warp-synchronous programming (below)
        // we need to declare our shared memory volatile so that the compiler
        // doesn't reorder stores to it and induce incorrect behavior.
		volatile T* smem = sdata;

        if (blockSize >=  64) { smem[tid] = mySum = mySum + smem[tid + 32]; }
        if (blockSize >=  32) { smem[tid] = mySum = mySum + smem[tid + 16]; }
        if (blockSize >=  16) { smem[tid] = mySum = mySum + smem[tid +  8]; }
        if (blockSize >=   8) { smem[tid] = mySum = mySum + smem[tid +  4]; }
        if (blockSize >=   4) { smem[tid] = mySum = mySum + smem[tid +  2]; }
        if (blockSize >=   2) { smem[tid] = mySum = mySum + smem[tid +  1]; }

	}

    // write result for this block to global mem
    if (tid == 0)
        *result = sdata[0];
}

/**
 * 	\brief		manages gpu data
 */
class weightSumData
{
public:
	static weightSumData* instance(int threadId, int N)
	{
		while ( weightSumData_.size() <= threadId ) // resize if needed
		{
			weightSumData_.push_back(NULL);
		}

		if( weightSumData_[threadId] == NULL ) // initialize if not already done
		{
			weightSumData_[threadId] = new weightSumData(N);
		}

		return weightSumData_[threadId];
	}

	void resize(int N)
	{
		std::cout << "Resize from " << nbElements_ << " to " << N << std::endl;
		hipDeviceSynchronize();
		hipMalloc((void**)&gpuWeights_, sizeof(DATA_TYPE) * N);
		hipMalloc((void**)&gpuRates_, sizeof(DATA_TYPE) * N);
		hipMalloc((void**)&gpuIdx_, sizeof(int) * N);

		nbElements_ = N;
	}

	DATA_TYPE* getRatePtr() { return gpuRates_; }
	DATA_TYPE* getWeightPtr() { return gpuWeights_; }
	int* getIndexPtr() { return gpuIdx_; }
	DATA_TYPE* getResultPtr() { return gpuResult_; }

private:
	weightSumData(int N)
	{
		gpuRates_ = NULL;
		gpuWeights_ = NULL;
		gpuResult_ = NULL;
		gpuIdx_ = NULL;
		nbElements_ = NULL;

		hipMalloc((void**)&gpuResult_, sizeof(DATA_TYPE));
		resize(N);
	}

	DATA_TYPE *gpuRates_;
	DATA_TYPE *gpuWeights_;
	DATA_TYPE *gpuResult_;
	int *gpuIdx_;
	int nbElements_;

	static std::vector<weightSumData*> weightSumData_;
};

std::vector<weightSumData*> weightSumData::weightSumData_ = std::vector<weightSumData*>();

DATA_TYPE weightedSum(std::vector<int> rank, std::vector<DATA_TYPE> value, std::vector<DATA_TYPE> preRates)
{
	int N = rank.size();
	int tId = omp_get_thread_num();

	double start1 = omp_get_wtime();
	hipMemcpy( weightSumData::instance(tId, N)->getWeightPtr(), value.data(), sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice );
	hipMemcpy( weightSumData::instance(tId, N)->getRatePtr(), preRates.data(), sizeof(DATA_TYPE) * N, hipMemcpyHostToDevice);
	hipMemcpy( weightSumData::instance(tId, N)->getIndexPtr(), rank.data(), sizeof(int) * N, hipMemcpyHostToDevice);
	std::cout << "Copying data ("<< N <<" synapses): "<< (omp_get_wtime() - start1)*1000.0 << " ms "<< std::endl;

	int numBlocks = (int)ceil(double(rank.size())/32.0);
	int smemSize = 64*sizeof(DATA_TYPE);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	std::cout << "Compute kernel ... "<< std::endl;
	hipEventRecord(start, 0);

	weightReduce<DATA_TYPE,32><<<numBlocks, 32, smemSize>>>(weightSumData::instance(tId, N)->getRatePtr(),
															weightSumData::instance(tId, N)->getWeightPtr(),
															weightSumData::instance(tId, N)->getIndexPtr(),
															N,
															weightSumData::instance(tId, N)->getResultPtr()
															);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime = 0.0;
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	std::cout << "Time for kernel ("<< rank.size() <<" synapses): "<< elapsedTime << " ms "<< std::endl;

	DATA_TYPE sum = 0.0;
	hipMemcpy(&sum, weightSumData::instance(tId, N)->getResultPtr(), sizeof(DATA_TYPE), hipMemcpyDeviceToHost);

	return sum;
}
